// Including Libraries
#include <stdio.h>
#include <hip/hip_runtime.h>

// Defining the CUDA Kernel
__global__ void vectorAdd(int **a, int **out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int x = i / n;
    if (x < n) {
        int y = i % n;
        out[x][y] = a[x][y] + 10;
    }
}


// Main Function
int main() {
    const int n = 5;

    int a[n][n], out[n][n];
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            a[i][j] = i * n + j;
        }
    }

    int **a_ptr, **out_ptr;
    int *a_rows[n], *out_rows[n];
    hipMalloc((void**)&a_ptr, n * sizeof(int*));
    hipMalloc((void**)&out_ptr, n * sizeof(int*));
    for (int i = 0; i < n; i++) {
        hipMalloc((void**)&a_rows[i], n * sizeof(int));
        hipMalloc((void**)&out_rows[i], n * sizeof(int));
    }

    hipMemcpy(a_ptr, a_rows, n * sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(out_ptr, out_rows, n * sizeof(int*), hipMemcpyHostToDevice);
    for (int i = 0; i < n; i++) {
        hipMemcpy(a_rows[i], a[i], n * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(out_rows[i], out[i], n * sizeof(int), hipMemcpyHostToDevice);
    }

    vectorAdd<<<4, 9>>>(a_ptr, out_ptr, n);

    for (int i = 0; i < n; i++) {
        hipMemcpy(out[i], out_rows[i], n * sizeof(int), hipMemcpyDeviceToHost);
    }

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d->%d ", a[i][j], out[i][j]);
        }
        printf("\n");
    }

    return 0;
}

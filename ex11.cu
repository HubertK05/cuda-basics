// Including Libraries
#include <stdio.h>
#include <hip/hip_runtime.h>

// Defining the CUDA Kernel
__global__ void convolution(int *a, int *b, int *out, int n, int m) {
    int block_size = blockDim.x;
    int i = threadIdx.x;
    for (int j = 0; j <= m; j++) {
        for (int k = i; k < n; k += block_size) {
            if (k >= j) out[k - j] += a[k] * b[j];
        }
        __syncthreads();
    }
}


// Main Function
int main() {
    const int n = 15;
    const int m = 4;
    int a[n], b[m], out[n];
    int *d_a, *d_b, *d_out;

    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_b, m * sizeof(int));
    hipMalloc((void**)&d_out, n * sizeof(int));

    for(int i = 0; i < n; i++) {
        a[i] = i;
        out[i] = 0;
    }
    for (int i = 0; i < m; i++) b[i] = i;
    
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, m * sizeof(int), hipMemcpyHostToDevice);

    convolution<<<1, 8>>>(d_a, d_b, d_out, n, m);

    hipMemcpy(out, d_out, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_out);

    for (int i = 0; i < n; i++) {
        printf("%d ", out[i]);
    }
    printf("\n");

    return 0;
}

// Including Libraries
#include <stdio.h>
#include <hip/hip_runtime.h>

// Defining the CUDA Kernel
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}


// Main Function
int main() {
    const int n = 256;
    int a[n], b[n], c[n];
    int *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_b, n * sizeof(int));
    hipMalloc((void**)&d_c, n * sizeof(int));

    // Initialize arrays and copy to device
    for(int i = 0; i < n; i++) {
        a[i] = b[i] = i;
    }
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    vectorAdd<<<1, n>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    for (int i = 0; i < n; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}

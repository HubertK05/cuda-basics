// Including Libraries
#include <stdio.h>
#include <hip/hip_runtime.h>

// Defining the CUDA Kernel
__global__ void vectorAdd(int *a, int *out, int n) {
    int i = threadIdx.x;
    __shared__ int shared[8];

    if (i < 8) {
        for (int j = max(0, i - 2); j <= i; j++) {
            shared[i] += a[j];
        }
    }
    __syncthreads();
    out[i] = shared[i];
}


// Main Function
int main() {
    const int n = 8;
    int a[n], out[n];
    int *d_a, *d_out;

    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_out, n * sizeof(int));

    for(int i = 0; i < n; i++) {
        a[i] = i;
    }
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);

    vectorAdd<<<1, 8>>>(d_a, d_out, n);

    hipMemcpy(out, d_out, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_out);

    for (int i = 0; i < n; i++) {
        printf("sum of last three: %d\n", out[i]);
    }

    return 0;
}

// Including Libraries
#include <stdio.h>
#include <hip/hip_runtime.h>

// Defining the CUDA Kernel
__global__ void vectorAdd(int *a, int *out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        out[i] = a[i] + 10;
    }
}


// Main Function
int main() {
    const int n = 9;
    int a[n], out[n];
    int *d_a, *d_out;

    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_out, n * sizeof(int));

    for(int i = 0; i < n; i++) {
        a[i] = i;
    }
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);

    vectorAdd<<<3, 4>>>(d_a, d_out, n);

    hipMemcpy(out, d_out, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_out);

    for (int i = 0; i < n; i++) {
        printf("%d + 10 = %d\n", a[i], out[i]);
    }

    return 0;
}

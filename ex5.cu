// Including Libraries
#include <stdio.h>
#include <hip/hip_runtime.h>

// Defining the CUDA Kernel
__global__ void vectorBroadcast(int *a, int *b, int **out, int n, int m) {
    int i = threadIdx.x / m;
    int j = threadIdx.x % m;
    if (i < n) {
        out[i][j] = a[j] + b[i];
    }
}


// Main Function
int main() {
    const int n = 4;
    const int m = 5;

    int a[m], b[n], out[n][m];
    
    for (int i = 0; i < m; i++) a[i] = i;
    for (int i = 0; i < n; i++) b[i] = 5 + i * 2;

    int *a_ptr, *b_ptr, **out_ptr;
    int *out_rows[n];
    hipMalloc((void**)&a_ptr, m * sizeof(int));
    hipMalloc((void**)&b_ptr, n * sizeof(int));
    hipMalloc((void**)&out_ptr, n * sizeof(int*));
    for (int i = 0; i < n; i++) {
        hipMalloc((void**)&out_rows[i], m * sizeof(int));
    }

    hipMemcpy(a_ptr, a, m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_ptr, b, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(out_ptr, out_rows, n * sizeof(int*), hipMemcpyHostToDevice);
    for (int i = 0; i < n; i++) {
        hipMemcpy(out_rows[i], out[i], m * sizeof(int), hipMemcpyHostToDevice);
    }

    vectorBroadcast<<<1, (n + 1) * (m + 1)>>>(a_ptr, b_ptr, out_ptr, n, m);

    for (int i = 0; i < n; i++) {
        hipMemcpy(out[i], out_rows[i], m * sizeof(int), hipMemcpyDeviceToHost);
    }

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("(%d, %d)->%d ", a[j], b[i], out[i][j]);
        }
        printf("\n");
    }

    return 0;
}

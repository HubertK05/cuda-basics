// Including Libraries
#include <stdio.h>
#include <hip/hip_runtime.h>

// Defining the CUDA Kernel
__global__ void vectorAdd(int *a, int *out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int local_i = threadIdx.x;
    
    __shared__ int shared[4];
    if (local_i < 4) {
        shared[i] = a[i] + 10;
    }
    __syncthreads();
    out[i] = shared[i];
}


// Main Function
int main() {
    const int n = 8;
    int a[n], out[n];
    int *d_a, *d_out;

    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_out, n * sizeof(int));

    for(int i = 0; i < n; i++) {
        a[i] = i;
    }
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);

    vectorAdd<<<2, 4>>>(d_a, d_out, n);

    hipMemcpy(out, d_out, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_out);

    for (int i = 0; i < n; i++) {
        printf("%d + 10 = %d\n", a[i], out[i]);
    }

    return 0;
}

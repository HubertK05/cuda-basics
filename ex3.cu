// Including Libraries
#include <stdio.h>
#include <hip/hip_runtime.h>

// Defining the CUDA Kernel
__global__ void vectorAdd(int *a, int *out, int n) {
    int i = threadIdx.x % n;
    if (i < n) {
        out[i] = a[i] + 10;
    }
}


// Main Function
int main() {
    const int n = 4;
    int a[n], out[n];
    int *d_a, *d_out;

    // Allocate device memory
    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_out, n * sizeof(int));

    // Initialize arrays and copy to device
    for(int i = 0; i < n; i++) {
        a[i] = i;
    }
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    vectorAdd<<<1, n * 2>>>(d_a, d_out, n);

    // Copy result back to host
    hipMemcpy(out, d_out, n * sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_a); hipFree(d_out);

    for (int i = 0; i < n; i++) {
        printf("%d + 10 = %d\n", a[i], out[i]);
    }

    return 0;
}
